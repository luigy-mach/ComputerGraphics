
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000
#define RADIUS 3
#define BLOCK_SIZE 32

__global__ void stencil(int *in, int *out) {

	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
 	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
 	int lindex = threadIdx.x + RADIUS;
 	// Read input elements into shared memory
 	temp[lindex] = in[gindex];

 	if (threadIdx.x < RADIUS) {
 	temp[lindex - RADIUS] = in[gindex - RADIUS];
 	temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
 	}

 	// Synchronize (ensure all the data is available)
 	__syncthreads();

 	// Apply the stencil
 	int result = 0;
 	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
 	result += temp[lindex + offset];
 	// Store the result
 	out[gindex] = result;
}


void fill_vec(int *a, int n){
	int i;
	for(i=1;i<=n;i++){
		//a[i]=rand()%99;
		a[i-1] = i;
	}
}

void print_vec(int *a, int n){
	int i;
	for(i=0;i<n;i++){
		printf("%d ",a[i]);
	}
	printf("\n");
}

int main()
{
	int *input,*output;
	int *d_input,*d_output;

	float time;

	int size = N * sizeof(int);

	input = (int*) malloc(size);
	output = (int*) malloc(size);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	fill_vec(input, N);
	
	hipMalloc((void **)&d_input, size);
	hipMalloc((void **)&d_output, size);
	
	//print_vec(input,N);

	hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

	hipEventRecord(start);
	stencil<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE,BLOCK_SIZE>>>(d_input, d_output);
	hipEventRecord(stop);

	hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

	hipEventElapsedTime(&time,start,stop);

	//print_vec(output,N);
	printf("Elapsed time: %f \n",time);
	return 0;
}
