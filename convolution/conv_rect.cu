
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAT_ROW 4
#define MAT_COL 6
#define CHANNELS 1
#define MASK_WIDTH  3
#define MASK_RADIUS MASK_WIDTH/2
#define O_TILE_WIDTH 12
#define BLOCK_WIDTH (O_TILE_WIDTH + (MASK_WIDTH-1))

void print_matrix(float* a,int n,int m)
{
        int i,j;
        for(i=0;i<n;i++)
        {
                for(j=0;j<m;j++)
                {
                        printf("%f ",a[i*m+j]);
                }
                printf("\n");
        }
}

void fill_mat(float* a,int n,int m)
{
        //srand(time(NULL));
        int i,j;
        for(i=0;i<n;i++)
        {
                for(j=0;j<m;j++)
                {
                        //a[i*n+j] = (rand()%2+1)*1.0;
                        a[i*m+j] = 1.0;
                }
        }
}

__global__ void convolution_shared(float *in, float* out,const float* __restrict__ M,int height, int width, int channels)
{

  float sum, pixel, maskVal;

  __shared__ float Ns[BLOCK_WIDTH][BLOCK_WIDTH];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row_o = blockIdx.y*O_TILE_WIDTH + ty;
  int col_o = blockIdx.x*O_TILE_WIDTH + tx;

  int row_i = row_o - MASK_RADIUS;
  int col_i = col_o - MASK_RADIUS;

  
  for (int c = 0; c < channels; c++) {

    if ( (row_i >= 0) && (row_i < height) &&
        (col_i >= 0) && (col_i < width) ) {
      Ns[ty][tx] = in[(row_i*width + col_i)*channels + c]; 
    }
    else {
      Ns[ty][tx] = 0.0f;
    }
    __syncthreads();

    sum = 0.0;
    if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {
      for (int y = 0; y < MASK_WIDTH; y++){
        for (int x = 0; x < MASK_WIDTH; x++){
          pixel = Ns[ty + y][tx + x];
          maskVal = M[y*MASK_WIDTH + x];
          sum += pixel*maskVal;
        }
      }
      if (row_o < height && col_o < width) {
        //out[ (row_o * width + col_o) * channels + c] = min(max(0.0f,sum),1.0f);
        out[ (row_o * width + col_o) * channels + c] = sum;
      }
    }
   // __syncthreads();
  }
}


int main()
{
        float *mat,*d_mat;
        float *mask,*d_mask;
        float *result,*d_result;

        //float elapsed_time=0;
        //cudaEvent_t start,stop;
        //cudaEventCreate(&start);
        //cudaEventCreate(&stop);

        int mat_size = MAT_ROW*MAT_COL*sizeof(float);
        int mask_size = MASK_WIDTH*MASK_WIDTH*sizeof(float);

        mat = (float*) malloc(mat_size);
        result = (float*) malloc(mat_size);
        mask = (float*) malloc(mask_size);

        fill_mat(mat,MAT_ROW,MAT_COL);
        fill_mat(mask,MASK_WIDTH,MASK_WIDTH);
        printf("Printing Matrix \n");
        print_matrix(mat,MAT_ROW,MAT_COL);
        printf("Printing Mask\n");
        print_matrix(mask,MASK_WIDTH,MASK_WIDTH);
        printf("\n");

        hipMalloc((void** )&d_mat,mat_size);
        hipMalloc((void** )&d_result,mat_size);
        hipMalloc((void** )&d_mask,mask_size);

        hipMemcpy(d_mat,mat,mat_size,hipMemcpyHostToDevice);
        hipMemcpy(d_mask,mask,mask_size,hipMemcpyHostToDevice);

        dim3 my_block(BLOCK_WIDTH,BLOCK_WIDTH);
        dim3 my_grid((MAT_COL + BLOCK_WIDTH-1)/my_block.x,(MAT_ROW + BLOCK_WIDTH-1)/my_block.y);

 		convolution_shared<<<my_grid,my_block>>>(d_mat, d_result, d_mask, MAT_ROW,MAT_COL,CHANNELS);
        hipMemcpy(result,d_result,mat_size,hipMemcpyDeviceToHost);
        printf("Printing result\n");
        print_matrix(result,MAT_ROW,MAT_COL);

        //printf("Elapsed time: %f\n",elapsed_time);
        return 0;
}

