// Luigy Machaca Arcana
// Computer science - Arequipa, Perú  2017



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <fstream>
#include <iostream>
#include <string>


using namespace std;
#define WIDTH_TILE 32



#define TAM_KERNEL 3
#define CONV_TILE_WIDTH 3
#define CONV_TILE_WIDTH2 CONV_TILE_WIDTH+TAM_KERNEL-1
//#define CONV_BLOCK_WIDTH 12

__global__ void convolution_shared(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){


	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	//const int row_tile = CONV_TILE_WIDTH+n_kernel_row-1;
	//const int col_tile = CONV_TILE_WIDTH+n_kernel_col-1;
	__shared__ int Ns[CONV_TILE_WIDTH2][CONV_TILE_WIDTH2];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int blockDimx = blockDim.x;
	int blockDimy = blockDim.y;


	int row = by*blockDimy + ty;
	int col = bx*blockDimx + tx;	


	if ( (row >= 0) && (row < n_rows_a) && (col >= 0) && (col < n_cols_a) ) 
	{
      Ns[ty][tx] = dd_mat_a[row][col]; 
    }
    else {
      Ns[ty][tx] = 0;
    }

    //int value;

	if( ((int)(n_kernel_row/2)-1)< row && row<(n_rows_a-(int)(n_kernel_row/2)) && 
		((int)(n_kernel_col/2)-1)< col && col<(n_cols_a-(int)(n_kernel_col/2)) 	){

		double offset = 0;
		for(int k=(n_kernel_row/2) ; k<CONV_TILE_WIDTH+CONV_TILE_WIDTH-(n_kernel_row/2); k++){
			for(int l=(n_kernel_col/2) ; l<CONV_TILE_WIDTH+CONV_TILE_WIDTH-(n_kernel_col/2) ; l++){
				double cc = dd_mat_b[k-n_kernel_row/2][l-n_kernel_col/2];
				double dd = 0;
				//dd = Ns[ty-(int)(n_kernel_row/2)+k][tx-(int)(n_kernel_col/2)+l];
				dd = Ns[k][l];
				offset += cc*dd;
			}
		}
		//dd_mat_c[row][col] = offset;
	}
		dd_mat_c[row][col] = -1;
}




__global__ void convolution(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int blockDimx = blockDim.x;
	int blockDimy = blockDim.y;


	int row = by*blockDimy + ty;
	int col = bx*blockDimx + tx;	

	if( ((int)(n_kernel_row/2)-1)< row && row<(n_rows_a-(int)(n_kernel_row/2)) && 
		((int)(n_kernel_col/2)-1)< col && col<(n_cols_a-(int)(n_kernel_col/2)) 	){

		double offset = 0;
		for(int k=0 ; k<n_kernel_row ; k++){
			for(int l=0 ; l<n_kernel_col ; l++){
				double cc = dd_mat_b[k][l];
				double dd = 0;
				dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
				/*
				if( (i-(int)(p/2)+k)>=0 && (j-(int)(q/2)+l)>=0 &&
					(i-(int)(p/2)+k)<n && (j-(int)(q/2)+l)<m  ){
					dd = in.at( i-(int)(p/2)+k , j-(int)(q/2)+l );
				}
				*/
				offset += cc*dd;
			}
		}
		dd_mat_c[row][col] = offset;
		//dd_mat_c[row][col] = -1;
	}

}


__global__ void convolution_complete(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int blockDimx = blockDim.x;
	int blockDimy = blockDim.y;


	int row = by*blockDimy + ty;
	int col = bx*blockDimx + tx;	


	if( row<n_rows_a && col<n_cols_a ){

		double offset = 0;
		for(int k=0 ; k<n_kernel_row ; k++){
			for(int l=0 ; l<n_kernel_col ; l++){
				double cc = dd_mat_b[k][l];
				double dd = 0;
				//dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
				if( (row-(int)(n_kernel_row/2)+k)>=0  && (row-(int)(n_kernel_row/2)+k)<n_rows_a &&
					(col-(int)(n_kernel_col/2)+l)>=0  && (col-(int)(n_kernel_col/2)+l)<n_cols_a  ){
					dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
					
				}
				offset += cc*dd;
			}
		}
		dd_mat_c[row][col] = offset;
		//dd_mat_c[row][col] = -1;
	}

}



__global__ void matrix_mult_shared(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	
	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int value = 0;

	int row = by*WIDTH_TILE + ty;
	int col = bx*WIDTH_TILE + tx;	

	int width = n_cols_a; //n_cols_a == n_rows_b

	int k;
	for( k=0 ; k<(int)(width-1+WIDTH_TILE)/(int)WIDTH_TILE ; ++k ){
		if (k*WIDTH_TILE+tx < n_cols_a && row < n_rows_a){
			Mds[ty][tx] = dd_mat_a[row][k*WIDTH_TILE+tx];
		}
        else{
			Mds[ty][tx] = 0;
        }

        if (k*WIDTH_TILE+ty < n_rows_b && col < n_cols_b){
			Nds[ty][tx] = dd_mat_b[k*WIDTH_TILE+ty][col];
        }
        else{
			Nds[ty][tx] = 0;
        }

		__syncthreads();
		int m;
		for(m=0 ; m<WIDTH_TILE ; ++m){
			value += Mds[ty][m]*Nds[m][tx];
		}
		__syncthreads();

	}

	if(row<n_rows_c && col<n_cols_c){
		dd_mat_c[row][col]=value;
	}
	

}


__global__ void matrix_mult(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){
	int value=0;


	int tx=threadIdx.x;
	int ty=threadIdx.y;


	int x = tx + blockIdx.x*blockDim.x;
	int y = ty + blockIdx.y*blockDim.y;

	if( y<n_rows_c && x<n_cols_c ){
		int i;
		for(i=0 ; i<n_cols_a ; i++){
			value += dd_mat_a[y][i] * dd_mat_b[i][x];
		}
		dd_mat_c[y][x]=value;
	} 
}


void fill_kernel_3x3(int** mat, int n, int m){
    mat[0][0]=2; mat[0][1]=3; mat[0][2]=2;
    mat[1][0]=2; mat[1][1]=0; mat[1][2]=2;
    mat[2][0]=2; mat[2][1]=3; mat[2][2]=2;
}



void fill(int** mat, int n, int m){
    srand(time(0));
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%5;
			//mat[i][j] = 1;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}



void create_copy(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols, int fillValue=-1){
	
	int i;

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}



void create(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols, int fillValue=-1){
	
	int i;
	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}
	if(fillValue==-1){
		fill(mat,n_rows,n_cols);	
	}
	else{
		fill_value(mat,n_rows,n_cols,fillValue);
	}

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}



void create_kernell_static(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols){
	
	int i;
	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}

	fill_kernel_3x3(mat,n_rows,n_cols); 

	int size_row = sizeof(int*) * n_rows;
	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}


void create_kernell_random(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols){
	
	int i,j; 

	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}

	srand(time(0));
	for(i=0; i<n_rows ;i++){
		for(j=0; j<n_cols ;j++)
			mat[i][j] = rand()%3+1;
			//mat[i][j] = 1;
	}

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}





int main(int argc, char *argv[]){

	

	/////////////////////////////////////////////////////
	

	int n_rows_a = 10;
	int n_cols_a = 10;

	int n_rows_b = 3;  //n_kernel
	int n_cols_b = 3;  //n_kernel

	int n_rows_c = 10;
	int n_cols_c = 10;



	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	//create_copy( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a	);
	create( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a	);
	
	create_kernell_static( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b 	); 
	//create_kernell_random( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b 	);

	create( mat_c, d_mat_c, dd_mat_c, n_rows_c, n_cols_c, 0	);



	/////////////////////////////////////////

	dim3 blockNum(WIDTH_TILE,WIDTH_TILE,1);
	dim3 grid((int)(n_cols_c-1+blockNum.x)/blockNum.x,(int)(n_rows_c-1+blockNum.y)/blockNum.y,1);
	printf("ty: %d, tx: %d\n",(int)(n_rows_c-1+blockNum.y)/blockNum.y, (int)(n_cols_c-1+blockNum.x)/blockNum.x);
	printf("grid_row: %d, grid_col: %d\n",grid.x , grid.y );

	////////////////////////////////////////////////////
	
	//convolution<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);
	convolution_shared<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//convolution_complete<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);



	//matrix_mult_shared<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//matrix_mult<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);
	

    /////////////////////////////////////////////////////

	hipMemcpy(mat_c[0],d_mat_c[0],sizeof(int)*n_rows_c*n_cols_c,hipMemcpyDeviceToHost);		
	
	
	printf("//////////////////\n");
	printf("//////////////////\n");
	print(mat_a,n_rows_a,n_cols_a);
	printf("//////////////////\n");
	print(mat_b,n_rows_b,n_cols_b);

	printf("//////////////////\n");
	print(mat_c,n_rows_c,n_cols_c);
	
	
  
	hipFree(dd_mat_a);
	hipFree(dd_mat_b);
	hipFree(dd_mat_c);
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
  	
  	free(mat_a);
  	free(mat_b);
  	free(mat_c);
  

	return 0;
}

