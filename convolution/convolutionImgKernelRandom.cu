// Luigy Machaca Arcana
// Computer science - Arequipa, Perú  2017



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <fstream>
#include <iostream>
#include <string>


using namespace std;


#define WIDTH_TILE 32


__global__ 
void convolution(int** dd_mat_a, int n_rows_a, int n_cols_a ,double** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;	

	if( ((int)(n_kernel_row/2)-1)< row && row<(n_rows_a-(int)(n_kernel_row/2)) && 
		((int)(n_kernel_col/2)-1)< col && col<(n_cols_a-(int)(n_kernel_col/2)) 	){

		double offset = 0;
		for(int k=0 ; k<n_kernel_row ; k++){
			for(int l=0 ; l<n_kernel_col ; l++){
				double cc = dd_mat_b[k][l];
				double dd = 0;
				dd = (double)dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
				offset += cc*dd;
			}
		}
		offset = offset>0?offset:0;
		dd_mat_c[row][col] = offset;
		//dd_mat_c[row][col] = dd_mat_a[row][col];
	}

}


__global__ 
void convolution_complete(int** dd_mat_a, int n_rows_a, int n_cols_a ,double** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;		

	if( row<n_rows_a && col<n_cols_a ){

		double offset = 0;
		for(int k=0 ; k<n_kernel_row ; k++){
			for(int l=0 ; l<n_kernel_col ; l++){
				double cc = dd_mat_b[k][l];
				double dd = 0;
				//dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
				if( (row-(int)(n_kernel_row/2)+k)>=0  && (row-(int)(n_kernel_row/2)+k)<n_rows_a &&
					(col-(int)(n_kernel_col/2)+l)>=0  && (col-(int)(n_kernel_col/2)+l)<n_cols_a  ){
					dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
					
				}
				offset += cc*dd;
			}
		}
		offset = -1/256*offset;
		offset = offset>0?offset:0;
		offset = (int)offset%255 + 1;
		dd_mat_c[row][col] = offset;
		//dd_mat_c[row][col] = -1;
	}

}



__global__ void matrix_mult_shared(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	
	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int value = 0;

	int row = by*WIDTH_TILE + ty;
	int col = bx*WIDTH_TILE + tx;	

	int width = n_cols_a; //n_cols_a == n_rows_b

	int k;
	for( k=0 ; k<(int)(width-1+WIDTH_TILE)/(int)WIDTH_TILE ; ++k ){
		if (k*WIDTH_TILE+tx < n_cols_a && row < n_rows_a){
			Mds[ty][tx] = dd_mat_a[row][k*WIDTH_TILE+tx];
		}
        else{
			Mds[ty][tx] = 0;
        }

        if (k*WIDTH_TILE+ty < n_rows_b && col < n_cols_b){
			Nds[ty][tx] = dd_mat_b[k*WIDTH_TILE+ty][col];
        }
        else{
			Nds[ty][tx] = 0;
        }

		__syncthreads();
		int m;
		for(m=0 ; m<WIDTH_TILE ; ++m){
			value += Mds[ty][m]*Nds[m][tx];
		}
		__syncthreads();

	}

	if(row<n_rows_c && col<n_cols_c){
		dd_mat_c[row][col]=value;
	}
	

}


__global__ void matrix_mult(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){
	int value=0;


	int tx=threadIdx.x;
	int ty=threadIdx.y;


	int x = tx + blockIdx.x*blockDim.x;
	int y = ty + blockIdx.y*blockDim.y;

	if( y<n_rows_c && x<n_cols_c ){
		int i;
		for(i=0 ; i<n_cols_a ; i++){
			value += dd_mat_a[y][i] * dd_mat_b[i][x];
		}
		dd_mat_c[y][x]=value;
	} 
}





void fill(int** mat, int n, int m){
    srand(time(0));
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			//mat[i][j] = rand()%3+1;
			mat[i][j] = 1;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}

void print2(double** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%f ",mat[i][j]);
		printf("\n");
	}
}

double max_value_matrix(int** mat,int n, int m){
	int i,j;
	int max = -100000;
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++){
			max = (mat[i][j] > max)?mat[i][j]:max;
		}
	}
	return max;
}

void normalize(int** mat,int n, int m, double value_normalice){
	int i,j;
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++){
			mat[i][j] = mat[i][j] / (double)value_normalice ;
		}
	}

}



void create_copy(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols){
	
	int i;

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}



void create(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols, int fillValue=-1){
	
	int i;
	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}
	if(fillValue==-1){
		fill(mat,n_rows,n_cols);	
	}
	else{
		fill_value(mat,n_rows,n_cols,fillValue);
	}

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}



void create_kernell_random(double**& mat, double**& d_mat, double**& dd_mat, int n_rows, int n_cols){
	

	int i,j;
	mat 	= (double** )malloc(sizeof(double*) * n_rows 			);	
	mat[0] 	= (double*  )malloc(sizeof(double ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}

	srand(time(0));
	for(i=0; i<n_rows ;i++){
		for(j=0; j<n_cols ;j++){
			mat[i][j] = (double)(rand()%100-50);
			//mat[i][j] = 1;
		}
	}


	int size_row = sizeof(double*) * n_rows;
	d_mat = (double**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(double) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(double) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );



}




void fill_kernel_3x3_1(double** mat, int n, int m, double scalar_kernel=1){
    mat[0][0]=0; mat[0][1]=	1; mat[0][2]=0;
    mat[1][0]=1; mat[1][1]=-4; mat[1][2]=1;
    mat[2][0]=0; mat[2][1]=	1; mat[2][2]=0;

    for(int i=0 ; i<n ; i++){
		for(int j=0 ; j<m ; j++){
			mat[i][j]=scalar_kernel*mat[i][j];
		}
	}
}

/////////////////////////////////////////////////////////////////////////
///////////////// Filter Sharpen
/////////////////////////////////////////////////////////////////////////

void fill_kernel_3x3_2(double** mat, int n, int m, double scalar_kernel=1){
			// 0  -1   0
			//-1   5  -1
			// 0  -1   0

    mat[0][0]=0; mat[0][1]=-1; mat[0][2]=0;
    mat[1][0]=-1; mat[1][1]=5; mat[1][2]=-1;
    mat[2][0]=0; mat[2][1]=-1; mat[2][2]=0;

    for(int i=0 ; i<n ; i++){
		for(int j=0 ; j<m ; j++){
			mat[i][j]=scalar_kernel*mat[i][j];
		}
	}
}

/////////////////////////////////////////////////////////////////////////
///////////////// Gaussian blur
/////////////////////////////////////////////////////////////////////////

void fill_kernel_5x5(double** mat, int n, int m, double scalar_kernel=1){
						// 1   4    6   4  1
						// 4  16   24  16  4
			//(-1/256)	// 6  24 -476  24  6
						// 4  16   24  16  4	
						// 1   4    6   4  1

	mat[0][0]=1; mat[0][1]=4 ; mat[0][2]=6   ; mat[0][3]=4 ; mat[0][4]=1;
	mat[1][0]=4; mat[1][1]=16; mat[1][2]=24  ; mat[1][3]=16; mat[1][4]=4;
	mat[2][0]=6; mat[2][1]=24; mat[2][2]=-476; mat[2][3]=24; mat[2][4]=6;
	mat[3][0]=4; mat[3][1]=16; mat[3][2]=24  ; mat[3][3]=16; mat[3][4]=4;
	mat[4][0]=1; mat[4][1]=4 ; mat[4][2]=6   ; mat[4][3]=4 ; mat[4][4]=1;

	printf("2222xxxxxxx %.25f\n",scalar_kernel);

	for(int i=0 ; i<n ; i++){
		for(int j=0 ; j<m ; j++){
			mat[i][j] = scalar_kernel*mat[i][j];
		}
	}
}


void create_kernell_static(double**& mat, double**& d_mat, double**& dd_mat, int n_rows, int n_cols, double scalar_kernel=1){
	
	int i;
	mat 	= (double** )malloc(sizeof(double*) * n_rows 		  );	
	mat[0] 	= (double*  )malloc(sizeof(double ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}

	//fill_kernel_3x3_1(mat,n_rows,n_cols, scalar_kernel); 
	fill_kernel_3x3_2(mat,n_rows,n_cols, scalar_kernel); 
	//fill_kernel_5x5(mat,n_rows,n_cols, scalar_kernel); 

	int size_row = sizeof(double*) * n_rows;
	d_mat = (double**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(double) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(double) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}


int main(int argc, char *argv[]){
	
	printf("//////////////////////////////////\n");
	char temp1[350];
	strcpy (temp1 , argv[1]);
	const char* img_input_name = temp1;

	char temp2[150];
	strcpy (temp2 , argv[1]);
	strcat (temp2 , ".out.random.kernel.random.pgm");
	const char* img_output_name = temp2;

	printf ("name in: %s\n",img_input_name);
	printf ("name out: %s\n",img_output_name);


	string title1,title2;
	char rows[15];
	char cols[15];
	char max_val[15];
	int n_rows = -1;
	int n_cols = -1;
	//int max_value = -1;

	/////////////////////////////////////////////////////////////

	ifstream myReadFile;
	myReadFile.open(img_input_name);

	char out_temp[100];
	
	int** mat_a;

	if (myReadFile.is_open()){

		std::getline(myReadFile,title1);
		std::getline(myReadFile,title2);

		myReadFile >> cols;
		n_cols = atoi(cols);
		//n_cols = 15;
		//cout << n_cols << endl;

		myReadFile >> rows;
		n_rows = atoi(rows);
		//n_rows = 15;
		//cout << n_rows << endl;


		myReadFile >> max_val;
		//max_value = atoi(max_val);
		//cout << max_value << endl;


		/////////////////////////////////////////////////////////////
		mat_a 		= (int** )malloc(sizeof(int*) * n_rows 			);	
		mat_a[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	
		
		for( int i=1 ; i<n_rows ; i++ ){
			mat_a[i] = mat_a[i-1]+n_cols;
		}

		/////////////////////////////////////////////////////////////
		int n_temp;
		for(int i=0 ; i<n_rows ; i++){
			for(int j=0 ; j<n_cols ; j++){
				if(!myReadFile.eof()){
					myReadFile >> out_temp;
					n_temp		 = atoi(out_temp);
					mat_a[i][j]	 = n_temp;
					//cout << n_temp << endl;	
				}
			}
		}
	}
	myReadFile.close();


	/////////////////////////////////////////////////////

		int n_rows_a = n_rows;
		int n_cols_a = n_cols;

		int n_rows_b = 3;  //n_kernel
		int n_cols_b = 3;  //n_kernel
	//double 	scalar_kernel = (-1)/(double)256; //escalar_kernel 
		//double 	scalar_kernel = 1; //escalar_kernel 
		//printf("escalar_kernel: %f\n",scalar_kernel);

		int n_rows_c = n_rows;
		int n_cols_c = n_cols;



	//int** mat_a; int** d_mat_a;	 int** dd_mat_a;	

	//int** mat_a;
				 		int** d_mat_a;	 	int** dd_mat_a;	
	double** mat_b;	 double** d_mat_b;	 double** dd_mat_b;	
	   int** mat_c;		int** d_mat_c;	 	int** dd_mat_c;	

	create_copy( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a);
	//create( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a	);
	
	//create_kernell_static( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b, scalar_kernel ); 
	create_kernell_random( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b );

	create( mat_c, d_mat_c, dd_mat_c, n_rows_c, n_cols_c, 0	);



	/////////////////////////////////////////

	dim3 blockNum(WIDTH_TILE,WIDTH_TILE,1);
	dim3 grid((int)(n_cols_c-1+blockNum.x)/blockNum.x,(int)(n_rows_c-1+blockNum.y)/blockNum.y,1);
	printf("ty: %d, tx: %d\n",(int)(n_rows_c-1+blockNum.y)/blockNum.y, (int)(n_cols_c-1+blockNum.x)/blockNum.x);
	printf("grid_row: %d, grid_col: %d\n",grid.x , grid.y );

	////////////////////////////////////////////////////
	
	convolution<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//convolution_complete<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);


	//matrix_mult_shared<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//matrix_mult<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);
	

    /////////////////////////////////////////////////////

	hipMemcpy(mat_c[0],d_mat_c[0],sizeof(int)*n_rows_c*n_cols_c,hipMemcpyDeviceToHost);		
	
	
	//printf("//////////////////////////////////\n");
	//printf("//////////////////////////////////\n");
	//print(mat_a,n_rows_a,n_cols_a);
	printf("//////// KERNELL RANDOM //////////\n");
	print2(mat_b,n_rows_b,n_cols_b);
	printf("//////////////////////////////////\n");
	//print(mat_c,n_rows_c,n_cols_c);
	
	


	//////////////////////////////////////////////

	double max_matrix = max_value_matrix(mat_c, n_rows_c, n_cols_c);


	//printf("<<<<<<<<<<<<<<<<<<<<<%f\n",max_matrix);


	ofstream myfile;
	myfile.open (img_output_name);
	myfile << title1 <<endl;
	myfile << title2 <<endl;
	myfile << n_cols_c <<" "<< n_rows_c <<endl;
	//myfile << max_value <<endl;
	myfile << max_matrix <<endl;

  	for(int i=0 ; i<n_rows_c ; i++){
		for(int j=0 ; j<n_cols_c ; j++){
			myfile << mat_c[i][j] <<endl;
		}
	}

	myfile.close();
	//////////////////////////////////////////////

  
	hipFree(dd_mat_a);
	hipFree(dd_mat_b);
	hipFree(dd_mat_c);
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
  	
  	free(mat_a);
  	free(mat_b);
  	free(mat_c);
  

	return 0;
}

