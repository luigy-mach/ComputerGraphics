// Luigy Machaca Arcana
// Computer science - Arequipa, Perú  2017



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <fstream>
#include <iostream>
#include <string>


using namespace std;




#define WIDTH_TILE 32



__global__ void convolution(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int blockDimx = blockDim.x;
	int blockDimy = blockDim.y;


	int row = by*blockDimy + ty;
	int col = bx*blockDimx + tx;	

	if( ((int)(n_kernel_row/2)-1)< row && row<(n_rows_a-(int)(n_kernel_row/2)) && 
		((int)(n_kernel_col/2)-1)< col && col<(n_cols_a-(int)(n_kernel_col/2)) 	){

		double offset = 0;
		for(int k=0 ; k<n_kernel_row ; k++){
			for(int l=0 ; l<n_kernel_col ; l++){
				double cc = dd_mat_b[k][l];
				double dd = 0;
				dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
				/*
				if( (i-(int)(p/2)+k)>=0 && (j-(int)(q/2)+l)>=0 &&
					(i-(int)(p/2)+k)<n && (j-(int)(q/2)+l)<m  ){
					dd = in.at( i-(int)(p/2)+k , j-(int)(q/2)+l );
				}
				*/
				offset += cc*dd;
			}
		}
		offset = offset>0?offset:0;
		offset = (int)offset%254 + 10;
		dd_mat_c[row][col] = offset;
		//dd_mat_c[row][col] = -1;
	}

}


__global__ void convolution_complete(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	int n_kernel_row = n_rows_b; //n_cols_b
	int n_kernel_col = n_cols_b; //n_cols_b
	
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int blockDimx = blockDim.x;
	int blockDimy = blockDim.y;


	int row = by*blockDimy + ty;
	int col = bx*blockDimx + tx;	


	if( row<n_rows_a && col<n_cols_a ){

		double offset = 0;
		for(int k=0 ; k<n_kernel_row ; k++){
			for(int l=0 ; l<n_kernel_col ; l++){
				double cc = dd_mat_b[k][l];
				double dd = 0;
				//dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
				if( (row-(int)(n_kernel_row/2)+k)>=0  && (row-(int)(n_kernel_row/2)+k)<n_rows_a &&
					(col-(int)(n_kernel_col/2)+l)>=0  && (col-(int)(n_kernel_col/2)+l)<n_cols_a  ){
					dd = dd_mat_a[row-(int)(n_kernel_row/2)+k][col-(int)(n_kernel_col/2)+l];
					
				}
				offset += cc*dd;
			}
		}
		offset = -1/256*offset;
		offset = offset>0?offset:0;
		offset = (int)offset%255 + 1;
		dd_mat_c[row][col] = offset;
		//dd_mat_c[row][col] = -1;
	}

}



__global__ void matrix_mult_shared(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	
	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int value = 0;

	int row = by*WIDTH_TILE + ty;
	int col = bx*WIDTH_TILE + tx;	

	int width = n_cols_a; //n_cols_a == n_rows_b

	int k;
	for( k=0 ; k<(int)(width-1+WIDTH_TILE)/(int)WIDTH_TILE ; ++k ){
		if (k*WIDTH_TILE+tx < n_cols_a && row < n_rows_a){
			Mds[ty][tx] = dd_mat_a[row][k*WIDTH_TILE+tx];
		}
        else{
			Mds[ty][tx] = 0;
        }

        if (k*WIDTH_TILE+ty < n_rows_b && col < n_cols_b){
			Nds[ty][tx] = dd_mat_b[k*WIDTH_TILE+ty][col];
        }
        else{
			Nds[ty][tx] = 0;
        }

		__syncthreads();
		int m;
		for(m=0 ; m<WIDTH_TILE ; ++m){
			value += Mds[ty][m]*Nds[m][tx];
		}
		__syncthreads();

	}

	if(row<n_rows_c && col<n_cols_c){
		dd_mat_c[row][col]=value;
	}
	

}


__global__ void matrix_mult(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){
	int value=0;


	int tx=threadIdx.x;
	int ty=threadIdx.y;


	int x = tx + blockIdx.x*blockDim.x;
	int y = ty + blockIdx.y*blockDim.y;

	if( y<n_rows_c && x<n_cols_c ){
		int i;
		for(i=0 ; i<n_cols_a ; i++){
			value += dd_mat_a[y][i] * dd_mat_b[i][x];
		}
		dd_mat_c[y][x]=value;
	} 
}


void fill_kernel_3x3(int** mat, int n, int m){
    mat[0][0]=-1; mat[0][1]=0; mat[0][2]=1;
    mat[1][0]=-1; mat[1][1]=0; mat[1][2]=1;
    mat[2][0]=-1; mat[2][1]=0; mat[2][2]=1;
}

void fill_kernel_5x5(int** mat, int n, int m){
//-1/256	// 1 4 6 4 1
		// 4 16 24 16 4
		// 6 24 -476 24 6
		// 4 16 24 16 4	
		// 1 4 6 4 1
	mat[0][0]=1; mat[0][1]=4 ; mat[0][2]=6   ; mat[0][3]=4 ; mat[0][4]=1;
	mat[1][0]=4; mat[1][1]=16; mat[1][2]=24  ; mat[1][3]=16; mat[1][4]=4;
	mat[2][0]=6; mat[2][1]=24; mat[2][2]=-476; mat[2][3]=24; mat[2][4]=6;
	mat[3][0]=4; mat[3][1]=16; mat[3][2]=24  ; mat[3][3]=16; mat[3][4]=4;
	mat[4][0]=1; mat[4][1]=4 ; mat[4][2]=6   ; mat[4][3]=4 ; mat[4][4]=1;

}



void fill(int** mat, int n, int m){
    srand(time(0));
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			//mat[i][j] = rand()%3+1;
			mat[i][j] = 1;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}



void create_copy(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols, int fillValue=-1){
	
	int i;

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}



void create(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols, int fillValue=-1){
	
	int i;
	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}
	if(fillValue==-1){
		fill(mat,n_rows,n_cols);	
	}
	else{
		fill_value(mat,n_rows,n_cols,fillValue);
	}

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}



void create_kernell_static(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols){
	
	int i;
	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}

	fill_kernel_3x3(mat,n_rows,n_cols); 
	//fill_kernel_5x5(mat,n_rows,n_cols); 

	int size_row = sizeof(int*) * n_rows;
	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}


void create_kernell_random(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols){
	
	int i,j; 

	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}

	srand(time(0));
	for(i=0; i<n_rows ;i++){
		for(j=0; j<n_cols ;j++)
			mat[i][j] = rand()%3+1;
			//mat[i][j] = 1;
	}

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}





int main(int argc, char *argv[]){

	

	string title1,title2;
	char rows[5];
	char cols[5];
	int n_rows = -1;
	int n_cols = -1;


	ifstream myReadFile;
	myReadFile.open("img/mario.pgm");

	char out_temp[10];
	
	int** mat_a;

	if (myReadFile.is_open()){

		std::getline(myReadFile,title1);
		std::getline(myReadFile,title2);

		myReadFile >> rows;
		n_rows = atoi(rows);
		//n_rows = 30;
		cout << n_rows << endl;

		myReadFile >> cols;
		n_cols = atoi(cols);
		//n_cols = 30;
		cout << n_cols << endl;

		/////////////////////////////////////////////////////////////
		mat_a 		= (int** )malloc(sizeof(int*) * n_rows 			);	
		mat_a[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	
		
		for( int i=1 ; i<n_rows ; i++ ){
			mat_a[i] = mat_a[i-1]+n_cols;
		}

		/////////////////////////////////////////////////////////////
		int n_temp;
		for(int i=0 ; i<n_rows ; i++){
			for(int j=0 ; j<n_cols ; j++){
				if(!myReadFile.eof()){
					myReadFile >> out_temp;
					n_temp = atoi(out_temp);
					mat_a[i][j] = n_temp;
					//cout << n_temp-1000 << endl;	
				}
			}
		}
		//while (!myReadFile.eof()){
		//	myReadFile >> out_temp;
		//	n_temp = atoi(out_temp);
		//	cout << n_temp-1000 << endl;
		//}
	}
	myReadFile.close();





	/////////////////////////////////////////////////////

	int n_rows_a = n_rows;
	int n_cols_a = n_cols;

	int n_rows_b = 5;  //n_kernel
	int n_cols_b = 5;  //n_kernel

	int n_rows_c = n_rows;
	int n_cols_c = n_cols;



	//int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	//int** mat_a;

				 int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	create_copy( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a	);
	//create( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a	);
	
	create_kernell_static( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b 	); 
	//create_kernell_random( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b 	);

	create( mat_c, d_mat_c, dd_mat_c, n_rows_c, n_cols_c, 0	);



	/////////////////////////////////////////

	dim3 blockNum(WIDTH_TILE,WIDTH_TILE,1);
	dim3 grid((int)(n_cols_c-1+blockNum.x)/blockNum.x,(int)(n_rows_c-1+blockNum.y)/blockNum.y,1);
	printf("ty: %d, tx: %d\n",(int)(n_rows_c-1+blockNum.y)/blockNum.y, (int)(n_cols_c-1+blockNum.x)/blockNum.x);
	printf("grid_row: %d, grid_col: %d\n",grid.x , grid.y );

	////////////////////////////////////////////////////
	
	//convolution<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	convolution_complete<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);



	//matrix_mult_shared<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//matrix_mult<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);
	

    /////////////////////////////////////////////////////

	hipMemcpy(mat_c[0],d_mat_c[0],sizeof(int)*n_rows_c*n_cols_c,hipMemcpyDeviceToHost);		
	
	
	printf("//////////////////\n");
	printf("//////////////////\n");
	print(mat_a,n_rows_a,n_cols_a);
	printf("//////////////////\n");
	print(mat_b,n_rows_b,n_cols_b);

	printf("//////////////////\n");
	print(mat_c,n_rows_c,n_cols_c);
	
	
	//////////////////////////////////////////////
	ofstream myfile;
	myfile.open ("mario3.pgm");
	myfile << title1 <<endl;
	myfile << title2 <<endl;
	myfile << n_rows_c <<" "<< n_cols_c <<endl;

  	for(int i=0 ; i<n_rows_c ; i++){
		for(int j=0 ; j<n_cols_c ; j++){
			myfile << mat_c[i][j] <<endl;
		}
	}

	myfile.close();
	//////////////////////////////////////////////

  
	hipFree(dd_mat_a);
	hipFree(dd_mat_b);
	hipFree(dd_mat_c);
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
  	
  	free(mat_a);
  	free(mat_b);
  	free(mat_c);
  

	return 0;
}

