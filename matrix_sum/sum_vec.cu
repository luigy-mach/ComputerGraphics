
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 20
#define M 3



__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < n )
		c[index] = a[index] + b[index];	

}

void random_ints(int *x, int n){
	int i;
	for(i=0;i<n;i++){
		x[i]=rand()%99;
	}
}

void print(int *a, int n){
	int i;
	for(i=0;i<n;i++){
		printf(" %d ",a[i]);
	}
	printf("\n");

}



int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	print(a,N);
	print(b,N);
	print(c,N);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	//add<<<N,1>>>(d_a, d_b, d_c);
	// Launch add() kernel on GPU with N threads
	//add<<<1,N>>>(d_a, d_b, d_c,N);
	add<<<(N + M-1) / M,M>>>(d_a, d_b, d_c, N);


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	printf("-------------------------------\n");
	print(c,N);
	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
